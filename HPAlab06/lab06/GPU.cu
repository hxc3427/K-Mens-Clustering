#include "hip/hip_runtime.h"
#include "common.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#include <iostream>

__constant__ Vector2 centroids[3];

// GPU Kernel to change the centroid
__global__ void KMeansKernel( Datapoint* data_d, long n, int k )
{
	// Retrieve our coordinates in the block
	long blockId = blockIdx.x + blockIdx.y * gridDim.x; 
	long threadId = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
if(threadId < n){
		for(int i=0; i<k; i++)
		{
			//change of centroids == false
			data_d[threadId].altered = false;

			//checks distance of i point from all 3 centroids.
			if(centroids[i].distSq(data_d[threadId].p) < centroids[data_d[threadId].cluster].distSq(data_d[threadId].p)){
				//check the point is in right centroid area
				if(data_d[threadId].cluster != i){
					data_d[threadId].cluster = i;
					//change of centroids == true
					data_d[threadId].altered = true;
				}
			}
		}
	}
}


bool KMeansGPU( Datapoint* data, long n, Vector2* clusters, int k ){

	// Error return value
	hipError_t status;

	// Number of bytes
	long bytes =  n * sizeof(Datapoint);

	// Pointers to the device
	Datapoint *data_d;

	// Allocate memory on the device
	status = hipMalloc((void**) &data_d, bytes);

	// Specify the size of the grid and the size of the block
	int tile= 512;
	int max_blocks = 65535;

	//threads in a block
	dim3 dimBlock(tile, 1);

	int gridx = 1;
	int gridy = 1;
	
	if(n/tile <= max_blocks)
		gridx = ceil((float)n/tile);
	else{
		gridx = max_blocks;
		gridy = ceil((float)n/(tile*max_blocks));
	}
	dim3 dimGrid(gridx, gridy);
	bool KMeans = true;

	// Copy the host input data to the device
		status = hipMemcpy(data_d, data, bytes, hipMemcpyHostToDevice);

	while(KMeans){

		KMeans=false;

		
		// Copy the host centroids data to the constant memory
		status = hipMemcpyToSymbol(HIP_SYMBOL(centroids), clusters, k*sizeof(Vector2), 0, hipMemcpyHostToDevice);
		
		// Launch the kernel
		KMeansKernel<<<dimGrid, dimBlock>>>(data_d, n, k);


		// Wait for completion
		hipDeviceSynchronize();		

	    // Retrieve the result
		hipMemcpy(data, data_d, bytes, hipMemcpyDeviceToHost);

		// Check for errors
         status = hipGetLastError();
		if (status != hipSuccess)
		{
			std::cout << "Kernel failed (data Memcpy) hipMemcpyDeviceToHost: " << hipGetErrorString(status) << 
							std::endl;
			hipFree(data_d);
			return false;
		}

// squential code to calculate new centroid
		//index for 3 clusters
	for (int j=0; j<k; j++){
		//count no. of elements under a centroid region
			long no_of_points=0;
			//index for no of points in space
			for(long i=0; i<n; i++)
			{
				if(data[i].cluster == j){
					//add x and y of each point in a particular cluster
					clusters[j].x += data[i].p.x;
					clusters[j].y += data[i].p.y;
					no_of_points++;
				}
				//if altered==true, set kmeans=true
				if(data[i].altered==true){
					KMeans=true;
				}
			}
			//new centroid
				clusters[j].x /= no_of_points;
				clusters[j].y /= no_of_points;
		}
	}

	hipFree(data_d);
	return true;
}